#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   : 
 To build use  : make
 ============================================================================
 */

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "./common/pgm.h"

const int degreeInc = 2;
const int degreeBins = 90;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;
//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  *acc = new int[rBins * 180 / degreeInc];
  memset (*acc, 0, sizeof (int) * rBins * 180 / degreeInc);
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++)  
    for (int j = 0; j < h; j++)
      {
        int idx = j * w + i;
        if (pic[idx] > 00)
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;  // y-coord has to be reversed
            float theta = 0;         // actual angle
            for (int tIdx = 0; tIdx < degreeBins; tIdx++)
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++;
                theta += radInc;
              }
          }
      }
}

//*****************************************************************
// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTran (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{
    // Calculate global ID
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;

  if (gloID >= w * h) return; // Check if gloID is within the bounds of the image size

  int xCent = w / 2;
  int yCent = h / 2;
  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  if (pic[gloID] > 0)
  {
    for (int tIdx = 0; tIdx < degreeBins; tIdx++)
    {
      float theta = tIdx * radInc;
      float r = xCoord * cos(theta) + yCoord * sin(theta);
      int rIdx = (r + rMax) / rScale;
      atomicAdd(&acc[rIdx * degreeBins + tIdx], 1);
    }
  }
}


//*****************************************************************
int main (int argc, char **argv)
{
  int i;

  PGMImage inImg (argv[1]);

  int *cpuht;
  int w = inImg.x_dim;
  int h = inImg.y_dim;

  // CPU calculation
  CPU_HoughTran (inImg.pixels, w, h, &cpuht);


  // compute values to be stored in device constant memory
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
    {
      pcCos[i] = cos (rad);
      pcSin[i] = sin (rad);
      rad += radInc;
    }

  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg.pixels; // h_in is just an alias here

  h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  // ! ========================================================================
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  // ! ========================================================================

  // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  int blockNum = ceil (w * h / 256);
  GPU_HoughTran <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale);

  hipDeviceSynchronize(); // Wait for the GPU launched work to complete

  // get results from device
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  // ! ========================================================================
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  // ! ========================================================================

  // compare CPU and GPU results
  for (i = 0; i < degreeBins * rBins; i++)
    {
      if (cpuht[i] != h_hough[i])
        printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
    }

  // ! ========================================================================
  printf("Kernel execution time: %f milliseconds\n", milliseconds);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  // ! ========================================================================

  // clean-up
  hipFree ((void *) d_in);
  hipFree ((void *) d_hough);
  free (h_hough);
  free (cpuht);
  free (pcCos);
  free (pcSin);
  hipDeviceReset ();

  return 0;
}